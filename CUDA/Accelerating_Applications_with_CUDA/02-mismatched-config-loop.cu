/*Exercise: Accelerating a For Loop with a Mismatched Execution Configuration
The program in 02-mismatched-config-loop.cu allocates memory, 
using cudaMallocManaged for a 1000 element array of integers, 
and then seeks to initialize all the values of the array in parallel using a CUDA kernel. 
This program assumes that both N and the number of threads_per_block are known. 
Your task is to complete the following two objectives, refer to the solution if you get stuck:

Assign a value to number_of_blocks that will make sure there are at least as many threads as there are elements in a to work on.
Update the initializeElementsTo kernel to make sure that it does not attempt to work on data elements that are out of range.
*/


#include <hip/hip_runtime.h>
#include <stdio.h>

/*
 * Currently, `initializeElementsTo`, if executed in a thread whose
 * `i` is calculated to be greater than `N`, will try to access a value
 * outside the range of `a`.
 *
 * Refactor the kernel definition to prevent out of range accesses.
 */

__global__ void initializeElementsTo(int initialValue, int *a, int N)
{
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  a[i] = initialValue;
}

int main()
{
  /*
   * Do not modify `N`.
   */

  int N = 1000;

  int *a;
  size_t size = N * sizeof(int);

  hipMallocManaged(&a, size);

  /*
   * Assume we have reason to want the number of threads
   * fixed at `256`: do not modify `threads_per_block`.
   */

  size_t threads_per_block = 256;

  /*
   * Assign a value to `number_of_blocks` that will
   * allow for a working execution configuration given
   * the fixed values for `N` and `threads_per_block`.
   */

  size_t number_of_blocks = 0;

  int initialValue = 6;

  initializeElementsTo<<<number_of_blocks, threads_per_block>>>(initialValue, a, N);
  hipDeviceSynchronize();

  /*
   * Check to make sure all values in `a`, were initialized.
   */

  for (int i = 0; i < N; ++i)
  {
    if(a[i] != initialValue)
    {
      printf("FAILURE: target value: %d\t a[%d]: %d\n", initialValue, i, a[i]);
      hipFree(a);
      exit(1);
    }
  }
  printf("SUCCESS!\n");

  hipFree(a);
}
