/*Exercise: Array Manipulation on both the Host and Device
The 01-double-elements.cu program allocates an array, initializes it with integer values on the host, 
attempts to double each of these values in parallel on the GPU, 
and then confirms whether or not the doubling operations were successful, on the host. 

Currently the program will not work: 
it is attempting to interact on both the host and the device with an array at pointer a, 
but has only allocated the array (using malloc) to be accessible on the host. 
Refactor the application to meet the following conditions

1. a should be available to both host and device code.
2. The memory at a should be correctly freed.

*/


#include <hip/hip_runtime.h>
#include <stdio.h>

/*
 * Initialize array values on the host.
 */

void init(int *a, int N)
{
  int i;
  for (i = 0; i < N; ++i)
  {
    a[i] = i;
  }
}

/*
 * Double elements in parallel on the GPU.
 */

__global__
void doubleElements(int *a, int N)
{
  int i;
  i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < N)
  {
    a[i] *= 2;
  }
}

/*
 * Check all elements have been doubled on the host.
 */

bool checkElementsAreDoubled(int *a, int N)
{
  int i;
  for (i = 0; i < N; ++i)
  {
    if (a[i] != i*2) return false;
  }
  return true;
}

int main()
{
  int N = 100;
  int *a;

  size_t size = N * sizeof(int);

  /*
   * Refactor this memory allocation to provide a pointer
   * `a` that can be used on both the host and the device.
   */

  a = (int *)malloc(size);

  init(a, N);

  size_t threads_per_block = 10;
  size_t number_of_blocks = 10;

  /*
   * This launch will not work until the pointer `a` is also
   * available to the device.
   */

  doubleElements<<<number_of_blocks, threads_per_block>>>(a, N);
  hipDeviceSynchronize();

  bool areDoubled = checkElementsAreDoubled(a, N);
  printf("All elements were doubled? %s\n", areDoubled ? "TRUE" : "FALSE");

  /*
   * Refactor to free memory that has been allocated to be
   * accessed by both the host and the device.
   */

  free(a);
}
